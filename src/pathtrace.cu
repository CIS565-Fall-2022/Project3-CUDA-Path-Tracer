#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>

#include "sceneStructs.h"
#include "material.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "mathUtil.h"
#include "sampler.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);

		// ACES tonemapping and gamma correction
		glm::vec3 color = image[index] / float(iter);
		glm::vec3 mapped = Math::ACES(color);
		mapped = color;
		mapped = Math::correctGamma(mapped);
		glm::ivec3 iColor = glm::clamp(glm::ivec3(mapped * 255.f), glm::ivec3(0), glm::ivec3(255));

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = iColor.x;
		pbo[index].y = iColor.y;
		pbo[index].z = iColor.z;
	}
}

#define PixelIdxForTerminated -1

static Scene* hstScene = nullptr;
static GuiDataContainer* guiData = nullptr;
static glm::vec3* devImage = nullptr;
static Geom* devGeoms = nullptr;
static Material* devMaterials = nullptr;
static PathSegment* devPaths = nullptr;
static PathSegment* devTerminatedPaths = nullptr;
static Intersection* devIntersections = nullptr;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static thrust::device_ptr<PathSegment> devPathsThr;
static thrust::device_ptr<PathSegment> devTerminatedPathsThr;
 
void InitDataContainer(GuiDataContainer* imGuiData) {
	guiData = imGuiData;
}

void pathTraceInit(Scene* scene) {
	hstScene = scene;

	const Camera& cam = hstScene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&devImage, pixelcount * sizeof(glm::vec3));
	hipMemset(devImage, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&devPaths, pixelcount * sizeof(PathSegment));
	hipMalloc(&devTerminatedPaths, pixelcount * sizeof(PathSegment));
	devPathsThr = thrust::device_ptr<PathSegment>(devPaths);
	devTerminatedPathsThr = thrust::device_ptr<PathSegment>(devTerminatedPaths);

	hipMalloc(&devGeoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(devGeoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&devMaterials, scene->materials.size() * sizeof(Material));
	hipMemcpy(devMaterials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&devIntersections, pixelcount * sizeof(Intersection));
	hipMemset(devIntersections, 0, pixelcount * sizeof(Intersection));

	// TODO: initialize any extra device memeory you need

	checkCUDAError("pathTraceInit");
}

void pathTraceFree() {

	hipFree(devImage);  // no-op if devImage is null
	hipFree(devPaths);
	hipFree(devTerminatedPaths);
	hipFree(devGeoms);
	hipFree(devMaterials);
	hipFree(devIntersections);
	// TODO: clean up any extra device memory you created

	checkCUDAError("pathTraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments) {

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		glm::vec4 r = sample4D(rng);

		PathSegment& segment = pathSegments[index];

		// Antialiasing and physically based camera (lens effect)

		float aspect = float(cam.resolution.x) / cam.resolution.y;
		float tanFovY = glm::tan(glm::radians(cam.fov.y));
		glm::vec2 pixelSize = 1.f / glm::vec2(cam.resolution);
		glm::vec2 scr = glm::vec2(x, y) * pixelSize;
		glm::vec2 ruv = scr + pixelSize * glm::vec2(r.x, r.y);
		ruv = 1.f - ruv * 2.f;

		glm::vec3 pLens = glm::vec3(Math::toConcentricDisk(r.z, r.w) * cam.lensRadius, 0.f);
		glm::vec3 pFocusPlane = glm::vec3(ruv * glm::vec2(aspect, 1.f) * cam.focalDist * tanFovY, cam.focalDist);
		glm::vec3 dir = pFocusPlane - pLens;
		dir = glm::normalize(glm::mat3(cam.right, cam.up, cam.view) * dir);

		segment.ray.origin = cam.position + cam.right * pLens.x + cam.up * pLens.y;
		segment.ray.direction = dir;

		segment.throughput = glm::vec3(1.f);
		segment.radiance = glm::vec3(0.f);
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth,
	int num_paths,
	PathSegment* pathSegments,
	Geom* geoms,
	int geoms_size,
	Intersection* intersections
) {
	int pathIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (pathIdx < num_paths) {
		PathSegment pathSegment = pathSegments[pathIdx];

		float dist;
		glm::vec3 intersectPoint;
		glm::vec3 normal;
		float tMin = FLT_MAX;
		int hitGeomIdx = -1;
		bool outside = true;

		glm::vec3 tmpIntersect;
		glm::vec3 tmpNormal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++) {
			Geom& geom = geoms[i];
			// TODO: add more intersection tests here... triangle? metaball? CSG?
			dist = intersectGeom(geom, pathSegment.ray, tmpIntersect, tmpNormal, outside);
			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (dist > 0.0f && tMin > dist) {
				tMin = dist;
				hitGeomIdx = i;
				intersectPoint = tmpIntersect;
				normal = tmpNormal;
			}
		}

		if (hitGeomIdx == -1) {
			intersections[pathIdx].dist = -1.0f;
		}
		else {
			//The ray hits something
			intersections[pathIdx].dist = tMin;
			intersections[pathIdx].materialId = geoms[hitGeomIdx].materialId;
			intersections[pathIdx].surfaceNormal = normal;
			intersections[pathIdx].position = intersectPoint;
			intersections[pathIdx].incomingDir = -pathSegment.ray.direction;
		}
	}
}

__global__ void pathIntegSampleSurface(
	int iter,
	PathSegment* segments,
	Intersection* intersections,
	Material* materials,
	int numPaths
) {
	const int SamplesConsumedOneIter = 10;

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= numPaths) {
		return;
	}
	Intersection intersec = intersections[idx];
	if (intersec.dist < 0) {
		// TODO
		// Environment map

		segments[idx].pixelIndex = PixelIdxForTerminated;
		return;
	}

	PathSegment& segment = segments[idx];
	thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 4 + iter * SamplesConsumedOneIter);
	Material material = materials[intersec.materialId];

	// TODO
	// Perform light area sampling and MIS

	if (material.type == Material::Type::Light) {
		// TODO
		// MIS

		segment.radiance += segment.throughput * material.baseColor * material.emittance;
		segment.remainingBounces = 0;
	}
	else {
		BSDFSample sample;
		materialSample(intersec.surfaceNormal, intersec.incomingDir, material, sample3D(rng), sample);

		if (sample.type == BSDFSampleType::Invalid) {
			// Terminate path if sampling fails
			segment.remainingBounces = 0;
		}
		else {
			bool isSampleDelta = (sample.type & BSDFSampleType::Specular);
			segment.throughput *= sample.bsdf / sample.pdf *
				(isSampleDelta ? 1.f : Math::absDot(intersec.surfaceNormal, sample.dir));
			segment.ray = makeOffsetedRay(intersec.position, sample.dir);
			segment.remainingBounces--;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths) {
		PathSegment iterationPath = iterationPaths[index];
		if (iterationPath.pixelIndex >= 0 && iterationPath.remainingBounces == 0) {
			image[iterationPath.pixelIndex] += iterationPath.radiance;
		}
	}
}

struct CompactTerminatedPaths {
	__host__ __device__ bool operator() (const PathSegment& segment) {
		return !(segment.pixelIndex >= 0 && segment.remainingBounces == 0);
	}
};

struct RemoveInvalidPaths {
	__host__ __device__ bool operator() (const PathSegment& segment) {
		return segment.pixelIndex < 0 || segment.remainingBounces == 0;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathTrace(uchar4* pbo, int frame, int iter) {

	const int traceDepth = hstScene->state.traceDepth;
	const Camera& cam = hstScene->state.camera;
	const int pixelCount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2D(8, 8);
	const dim3 blocksPerGrid2D(
		(cam.resolution.x + blockSize2D.x - 1) / blockSize2D.x,
		(cam.resolution.y + blockSize2D.y - 1) / blockSize2D.y);

	// 1D block for path tracing
	const int blockSize1D = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera<<<blocksPerGrid2D, blockSize2D>>>(cam, iter, traceDepth, devPaths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	int numPaths = pixelCount;

	auto devTerminatedThr = devTerminatedPathsThr;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {
		// clean shading chunks
		hipMemset(devIntersections, 0, pixelCount * sizeof(Intersection));

		// tracing
		dim3 numBlocksPathSegmentTracing = (numPaths + blockSize1D - 1) / blockSize1D;
		computeIntersections<<<numBlocksPathSegmentTracing, blockSize1D>>>(
			depth, 
			numPaths,
			devPaths, 
			devGeoms,
			hstScene->geoms.size(), 
			devIntersections
		);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		pathIntegSampleSurface<<<numBlocksPathSegmentTracing, blockSize1D>>>(
			iter, devPaths, devIntersections, devMaterials, numPaths
		);

		// Compact paths that are terminated but carry contribution into a separate buffer
		devTerminatedThr = thrust::remove_copy_if(devPathsThr, devPathsThr + numPaths, devTerminatedThr, CompactTerminatedPaths());
		// Only keep active paths
		auto end = thrust::remove_if(devPathsThr, devPathsThr + numPaths, RemoveInvalidPaths());
		numPaths = end - devPathsThr;
		//std::cout << "Remaining paths: " << numPaths << "\n";

		iterationComplete = numPaths == 0;

		if (guiData != nullptr) {
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelCount + blockSize1D - 1) / blockSize1D;
	int numContributing = devTerminatedThr.get() - devTerminatedPaths;
	finalGather<<<numBlocksPixels, blockSize1D>>>(numContributing, devImage, devTerminatedPaths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2D, blockSize2D>>>(pbo, cam.resolution, iter, devImage);

	// Retrieve image from GPU
	hipMemcpy(hstScene->state.image.data(), devImage,
		pixelCount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathTrace");
}