#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define SORT_BY_MATERIAL 1
#define CACHE_FIRST_BOUNCE 0
#define ANTI_ALIASING 0

#define DOF 0
#define LENS_RADIUS 0.4
#define FOCAL_DISTANCE 12.0

#define MOTION_BLUR 1

#define BOUNDINGBOX 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
static ShadeableIntersection* dev_intersections_cache = NULL;
static Triangle* dev_triangles = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	// if geoms contain mesh, allocate dev_triangles for it
	if (scene->hasMesh && scene->meshGeomId != -1) {
		Geom mesh = scene->geoms[scene->meshGeomId];
		hipMalloc(&dev_triangles, mesh.numOfTriangles * sizeof(Triangle));
		hipMemcpy(dev_triangles, mesh.triangles, mesh.numOfTriangles * sizeof(Triangle), hipMemcpyHostToDevice);
	}

#if CACHE_FIRST_BOUNCE
	hipMalloc(&dev_intersections_cache, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections_cache, 0, pixelcount * sizeof(ShadeableIntersection));
#endif

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_triangles);
#if CACHE_FIRST_BOUNCE
	hipFree(dev_intersections_cache);
#endif

	checkCUDAError("pathtraceFree");
}


__host__ __device__
glm::vec3 concentricSampleDisk(const glm::vec2& sampler) {
	float x = sampler.x;
	float y = sampler.y;
	float phi, r;
	float a = 2 * x - 1.f;
	float b = 2 * y - 1.f;

	if (a > -b) {
		if (a > b) {
			r = a;
			phi = (PI / 4) * (b / a);
		}
		else {
			r = b;
			phi = (PI / 4) * (2 - (a / b));
		}
	}
	else {
		if (a < b) {
			r = -a;
			phi = (PI / 4) * (4 + (b / a));
		}
		else {
			r = -b;
			if (b < 0 || b > 0) {
				phi = (PI / 4) * (6 - (a / b));
			}
			else {
				phi = 0;
			}
		}
	}
	return glm::vec3(cosf(phi) * r, sinf(phi) * r, 0);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
		
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, x, y);
		thrust::uniform_real_distribution<float> u01(0, 1);

#if ANTI_ALIASING
		
		x += u01(rng) * 2.0;
		y += u01(rng) * 2.0;
#endif
		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#if DOF
		/*glm::vec3 lens = concentricSampleDisk(glm::vec2(u01(rng), u01(rng))) * (float)LENS_RADIUS;
		glm::vec3 point = segment.ray.origin + lens;
		glm::vec3 pFocus = segment.ray.origin + (float)FOCAL_DISTANCE * segment.ray.direction;

		segment.ray.origin = point;
		segment.ray.direction = glm::normalize(pFocus - point);*/

		//Sample point on lens
		glm::vec3 point = concentricSampleDisk(glm::vec2(u01(rng), u01(rng))) * (float)LENS_RADIUS;

		glm::vec3 ref = cam.position + (cam.view * (float)FOCAL_DISTANCE);

		float aspect = ((float)cam.resolution.x / (float)cam.resolution.y);
		float angle = glm::radians(cam.fov.y);
		glm::vec3 V = cam.up * (float)FOCAL_DISTANCE * tan(angle);
		glm::vec3 H = cam.right * (float)FOCAL_DISTANCE * aspect * tan(angle);

		float ndc_x = 1.f - ((float)x / cam.resolution.x) * 2.f;
		float ndc_y = 1.f - ((float)y / cam.resolution.y) * 2.f;

		//Compute point on plane of focus
		glm::vec3 pFocus = ref + ndc_x * H + ndc_y * V;

		//Update ray for effect of lens
		segment.ray.origin = cam.position + (cam.up * point.y) + (cam.right * point.x);
		segment.ray.direction = glm::normalize(pFocus - segment.ray.origin);
#endif
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, Triangle* triangles
	, int geoms_size
	, ShadeableIntersection* intersections
	, int iter
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				if (MOTION_BLUR && geom.materialid == 8) {
					thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, 0);
					thrust::uniform_real_distribution<float> u01(0, 1);
					//Jitter the ray randomly about any axes 
					Ray jittered = pathSegment.ray;
					jittered.origin += u01(rng) * glm::vec3(0.75f, 0.75f, 0.f);
					t = sphereIntersectionTest(geom, jittered, tmp_intersect, tmp_normal, outside);
				}
				else {
					t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				}
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?
			else if (geom.type == MESH) {
#if BOUNDINGBOX
				// If intersect with the boundingbox of the obj, then compute the actual intersection point
				if (boundingBoxIntersectionTest(geom.boundingbBox, pathSegment.ray)) {
					t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, triangles, outside);
				}
#else
				t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, triangles, outside);
#endif
			}
			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        // if the intersection exists...
        if (intersection.t > 0.0f) 
        {  
            // Set up the RNG
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If hit light
            if (material.emittance > 0.0f) {
            pathSegments[idx].color *= (materialColor * material.emittance);
            pathSegments[idx].remainingBounces = 0;
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                pathSegments[idx].remainingBounces -= 1;
                // multiply the rayColor by the material color
                pathSegments[idx].color *= materialColor;
                glm::vec3 isect = getPointOnRay(pathSegments[idx].ray, intersection.t);
                scatterRay(pathSegments[idx], isect, intersection.surfaceNormal, material, rng);
          
            }
        } 
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

struct is_reach_max_depth
{
    __host__ __device__
        bool operator()(const PathSegment &path)
    {
        return path.remainingBounces != 0;
    }
};

struct sort_isect
{
    __host__ __device__
        bool operator()(const ShadeableIntersection& isect1, const ShadeableIntersection& isect2)
    {
        return isect1.materialId < isect2.materialId;
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete) {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

		// Use the cached first bounce
		if (CACHE_FIRST_BOUNCE && !ANTI_ALIASING && depth == 0 && iter != 1) {
			hipMemcpy(dev_intersections, dev_intersections_cache, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		
		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, dev_triangles
				, hst_scene->geoms.size()
				, dev_intersections
				, iter
				);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();

			// Cache first bounce
			if (CACHE_FIRST_BOUNCE && !ANTI_ALIASING && depth == 0 && iter == 1) {
				hipMemcpy(dev_intersections_cache, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
		}
        depth++;

#if SORT_BY_MATERIAL
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, sort_isect());
#endif
        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
        iter,
        num_paths,
        dev_intersections,
        dev_paths,
        dev_materials
        );

        // stream compaction
        dev_path_end = thrust::partition(thrust::device, dev_paths, dev_path_end, is_reach_max_depth());
        num_paths = dev_path_end - dev_paths;
        if (num_paths == 0) {
            iterationComplete = true;
        }
        else iterationComplete = false;

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
