#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/partition.h>
#include <thrust/random.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

// impl switches
#define COMPACTION
// #define FAKE_SHADE

void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#ifndef NDEBUG
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(int iter, glm::vec3* pixs, uchar4* pbo, glm::ivec2 resolution) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = pixs[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;

static glm::vec3*             dev_image;
static Geom*                  dev_geoms;
static Material*              dev_materials;
static PathSegment*           dev_paths;
static ShadeableIntersection* dev_intersections;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	ALLOC(dev_image, pixelcount);
	ZERO(dev_image, pixelcount);

	ALLOC(dev_paths, pixelcount);

	ALLOC(dev_geoms, scene->geoms.size());
	H2D(dev_geoms, scene->geoms.data(), scene->geoms.size());
	
	ALLOC(dev_materials, scene->materials.size());
	H2D(dev_materials, scene->materials.data(), scene->materials.size());
	
	ALLOC(dev_intersections, pixelcount);
	ZERO(dev_intersections, pixelcount);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {

	FREE(dev_image);
	FREE(dev_paths);
	FREE(dev_geoms);
	FREE(dev_materials);
	FREE(dev_intersections);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		Ray cam_ray;
		cam_ray.origin = cam.position;
		
		// TODO: implement antialiasing by jittering the ray
		cam_ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		PathSegment& segment = pathSegments[index];
		segment.init(traceDepth, index, cam_ray);
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth,
	int num_paths,
	PathSegment* pathSegments,
	int geoms_size,
	Geom* geoms,
	ShadeableIntersection* intersections)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (path_index >= num_paths) {
		return;
	}
	PathSegment pathSegment = pathSegments[path_index];

#ifndef COMPACTION
	if (!pathSegment.remainingBounces) {
		return;
	}
#endif // COMPACTION

	assert(pathSegment.remainingBounces > 0);

	float t;
	glm::vec3 intersect_point;
	glm::vec3 normal;
	float t_min = FLT_MAX;
	int hit_geom_index = -1;
	bool outside = true;

	glm::vec3 tmp_intersect;
	glm::vec3 tmp_normal;

	// naive parse through global geoms

	for (int i = 0; i < geoms_size; i++) {
		Geom& geom = geoms[i];

		if (geom.type == CUBE)
		{
			t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
		} else if (geom.type == SPHERE)
		{
			t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
		}
		// TODO: add more intersection tests here... triangle? metaball? CSG?

		// Compute the minimum t from the intersection tests to determine what
		// scene geometry object was hit first.
		if (t > 0.0f && t_min > t)
		{
			t_min = t;
			hit_geom_index = i;
			intersect_point = tmp_intersect;
			normal = tmp_normal;
		}
	}

	if (hit_geom_index == -1)
	{
		intersections[path_index].t = -1.0f;
	} else
	{
		//The ray hits something
		intersections[path_index].t = t_min;
		intersections[path_index].materialId = geoms[hit_geom_index].materialid;
		intersections[path_index].surfaceNormal = normal;
	}
}

__global__ void shadeMaterial(
    int iter,
	int num_paths,
	PathSegment* paths,
	ShadeableIntersection* shadeableIntersections,
	Material* materials) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_paths) {
		return;
	}

	PathSegment& path = paths[idx];
	ShadeableIntersection intersection = shadeableIntersections[idx];

#ifndef COMPACTION
	if (!path.remainingBounces) {
		return;
	}
#endif // COMPACTION

	assert(path.remainingBounces > 0);

	if (intersection.t > 0.0f) {
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

		Material material = materials[intersection.materialId];
		glm::vec3 materialColor = material.color;

		// If the material indicates that the object was a light, "light" the ray
		if (material.emittance > 0.0f) {
			path.color *= (materialColor * material.emittance);
			path.terminate();
		} else {
			glm::vec3 hit = intersection.t * path.ray.direction + path.ray.origin;
			scatterRay(path, hit, glm::normalize(intersection.surfaceNormal), material, rng);
		}
	} else {
		path.color = BACKGROUND_COLOR;
		path.terminate();
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, PathSegment* paths
	, ShadeableIntersection* shadeableIntersections
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		PathSegment& path = paths[idx];
		ShadeableIntersection intersection = shadeableIntersections[idx];

		assert(path.remainingBounces > 0);

		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				path.color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				path.color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				path.color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		} else {
			path.color = BACKGROUND_COLOR;
		}

		path.terminate();
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int numPixels, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < numPixels) {
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int const frame, int const iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera KERN_PARAM(blocksPerGrid2d, blockSize2d) (cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks
	for (int depth = 0, num_paths = pixelcount; num_paths > 0 && depth < traceDepth; ++depth) {
		
		// clean shading chunks
		MEMSET(dev_intersections, 0, num_paths);

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections KERN_PARAM(numblocksPathSegmentTracing, blockSize1d) (
			depth,
			num_paths,
			dev_paths,
			hst_scene->geoms.size(),
			dev_geoms,
			dev_intersections
		);

		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.
#ifdef SORT_MAT

#endif

#ifdef FAKE_SHADE
#define shadeMaterial shadeFakeMaterial
#endif
		shadeMaterial KERN_PARAM(numblocksPathSegmentTracing, blockSize1d) (
			iter,
			num_paths,
			dev_paths,
			dev_intersections,
			dev_materials
		);

		checkCUDAError("shadeMaterial");
		hipDeviceSynchronize();

#ifdef COMPACTION
		auto ptr = thrust::device_ptr<PathSegment>(dev_paths);
		num_paths = thrust::partition(ptr, ptr + num_paths, PathSegment::Pred()) - ptr;
#endif // COMPACTION

#ifdef MAX_DEPTH_OVERRIDE
		if (depth == MAX_DEPTH_OVERRIDE)
			break;
#endif
	}
	
	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather KERN_PARAM(numBlocksPixels, blockSize1d) (pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO KERN_PARAM(blocksPerGrid2d, blockSize2d) (iter, dev_image, pbo, cam.resolution);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}