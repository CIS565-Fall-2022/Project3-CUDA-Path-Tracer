#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/functional.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "thrust/device_vector.h"
#include "thrust/remove.h"
#include "thrust/execution_policy.h"
#include "thrust/sort.h"


#define ERRORCHECK 1

//integrator
#define DIRECTLIGHTING 1
#define FULLLIGHTING 0

//method
#define MISSAMPLING 0   //for direct light integrator

//optimize
#define SORTBYMATERIAL 0
#define CACHEFIRSTBOUNCE 1


#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
static ShadeableIntersection* dev_firstIntersections = NULL;
static Geom* dev_lights = NULL;
static PathSegment* dev_paths2 = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceInit(Scene* scene) {
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_firstIntersections, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_lights, scene->lights.size() * sizeof(Geom));
	hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Geom), hipMemcpyHostToDevice);

#if MISSAMPLING
	hipMalloc(&dev_paths2, pixelcount * sizeof(PathSegment));
#endif

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_firstIntersections);
	hipFree(dev_lights);
#if MISSAMPLING
	hipFree(dev_paths2);
#endif
	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(0.0f); 
		segment.beta = glm::vec3(1.f);
		segment.lightGeomId = -1;  //the light been chosen for this path

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; ++i)
		{
			//get every object in the scene, and do a intersection test for every one of them.
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SQUARE_PLANE) {
				t = squarePlaneIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
			intersections[path_index].materialId = INT_MAX;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].geomId = hit_geom_index;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
	, float& pdf_f_f
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			// here we assume the ray stops when it hit anything with emittance bigger than 0.f
			if (material.emittance > 0.0f) {
				pathSegments[idx].color += pathSegments[idx].beta * (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				glm::vec3 intersect = pathSegments[idx].ray.origin + intersection.t * pathSegments[idx].ray.direction;
				scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng, pdf_f_f);
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
* Used for stream compaction
*/
struct isTerminate {
	__host__ __device__
		bool operator()(const ShadeableIntersection& intersection) {
		return intersection.t > -1.0;
	}
};

/**
* Used for material sort
*/
struct compareMaterialId {
	__host__ __device__
		bool operator()(const ShadeableIntersection& isectA, const ShadeableIntersection& isectB) {
		return isectA.materialId < isectB.materialId;
	}
};

struct remainingBounceIsNot0 {
	__host__ __device__
		bool operator()(const PathSegment& p1) {
		return (p1.remainingBounces > 0);
	}
};

struct comparePixelIdx {
	__host__ __device__
		bool operator()(const PathSegment& p1, const PathSegment& p2) {
		return (p1.pixelIndex < p2.pixelIndex);
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 * 
 * @param iter: number of interation from runCuda()
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;   //the depth of each ray
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;
	const int num_lights = hst_scene->lights.size();

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);   //64 threads per block
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing
	
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
#if MISSAMPLING
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths2);
#endif
	checkCUDAError("generate camera ray");

#if FULLLIGHTING

#elif DIRECTLIGHTING
	directLightIntegrator(iter, pixelcount, blockSize1d, num_lights);
#else
	naiveIntegrator(iter, pixelcount, traceDepth, blockSize1d);
#endif


	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}

void naiveIntegrator(int iter, 
	int pixelcount, 
	int traceDepth, 
	int blockSize1d) 
{
	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = pixelcount;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) {

		// clean shading chunks(intersections)
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if CACHEFIRSTBOUNCE
		//iter starts with 1
		if (iter > 1 && depth == 0) {
			hipMemcpy(dev_intersections, dev_firstIntersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else {
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				);
			hipDeviceSynchronize();
			if (iter == 1 && depth == 0) {
				hipMemcpy(dev_firstIntersections, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			}
		}
#else
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			);
#endif
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		++depth;

		// sort by material
#if SORTBYMATERIAL
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, compareMaterialId());
#endif

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
	  // evaluating the BSDF.
	  // Start off with just a big kernel that handles all the different
	  // materials you have in the scenefile.
	  // TODO: compare between directly shading the path segments and shading
	  // path segments that have been reshuffled to be contiguous in memory.
		float pdf_f_f;
		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			pdf_f_f
			);

		//remove_if return the index(address) of the first wrong element
		//can't use remove_if, since I dont want these paths to disappear.
		/*dev_path_end = thrust::remove_if(thrust::device, dev_paths, dev_paths + num_paths, remainingBounceIs0());
		num_paths = dev_path_end - dev_paths;*/
		//use partition here
		dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, remainingBounceIsNot0());
		num_paths = dev_path_end - dev_paths;

		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
		if (num_paths == 0 || traceDepth == depth) {
			iterationComplete = true;
		}
	}
}


/**
* sample a direction, update beta
*/
__global__ void shadeMaterialDirectLight(
	const int iter
	, const int num_paths
	, const ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, const Material* materials
	, const Geom* dev_lights
	, const int num_lights
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...

			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			glm::vec3 intersect = pathSegments[idx].ray.origin + intersection.t * pathSegments[idx].ray.direction;

			if (material.emittance > 0.0f) {
				pathSegments[idx].color += pathSegments[idx].beta * (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				//randomly choose a light source and a point on that light source to produce a ray
				//and update pathsegment's ray direction, origin and beta
				float pdf_l_l;
				float pdf_l_f;
				scatterRayToLight(
					pathSegments[idx],
					intersect,
					intersection.surfaceNormal,
					material,
					rng,
					pathSegments[idx].lightGeomId,
					dev_lights,
					num_lights,
					pdf_l_l,
					pdf_l_f
				);
			}
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

__global__ void shadeLastDirLight(
	const ShadeableIntersection* shadeableIntersections,
	PathSegment* pathSegments,
	const Material* materials,
	const int num_paths
) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths) {
		ShadeableIntersection intersection = shadeableIntersections[idx];
		PathSegment& pathSegment = pathSegments[idx];
		if (intersection.geomId == pathSegment.lightGeomId) {
			//it hits the right light!!!
			if (intersection.materialId == INT_MAX) {
				pathSegment.color = glm::vec3(0.f);
			}
			else {
				Material material = materials[intersection.materialId];
				pathSegment.color += (pathSegment.beta * (material.color * material.emittance));
			}
		}
		else {
			pathSegment.color = glm::vec3(0.f);
		}
	}
}

void directLightIntegrator(int iter, int pixelcount, int blockSize1d, int num_lights) {
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = pixelcount;
	
	//depth 0
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	if (iter > 1) {
		hipMemcpy(dev_intersections, dev_firstIntersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
	}
	else {
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			);
			hipMemcpy(dev_firstIntersections, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
	}
#if SORTBYMATERIAL
	thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, compareMaterialId());
#endif
	//shade and choose a vec from intersect to light
	shadeMaterialDirectLight<<<numblocksPathSegmentTracing, blockSize1d >>>(
		iter
		, num_paths
		, dev_intersections
		, dev_paths
		, dev_materials
		, dev_lights
		, num_lights
	);
	dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, remainingBounceIsNot0());
	num_paths = dev_path_end - dev_paths;


	//depth 2
	numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	if (guiData != NULL)
	{
		guiData->TracedDepth = 1;
	}

	computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
		num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);

	shadeLastDirLight << < numblocksPathSegmentTracing, blockSize1d >> > (
		dev_intersections,
		dev_paths,
		dev_materials,
		num_paths
	);

	if (guiData != NULL)
	{
		guiData->TracedDepth = 2;
	}
}