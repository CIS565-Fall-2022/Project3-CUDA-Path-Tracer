#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            thrust::host_vector<int> host_data(n);
            for (int i = 0; i < n; i++)
            {
                host_data[i] = idata[i];
            }

            thrust::device_vector<int> dev_data = host_data;
            timer().startGpuTimer();
            thrust::exclusive_scan(dev_data.begin(), dev_data.end(), dev_data.begin());
            timer().endGpuTimer();

            thrust::copy(dev_data.begin(), dev_data.end(), odata);
        }
    }
}
