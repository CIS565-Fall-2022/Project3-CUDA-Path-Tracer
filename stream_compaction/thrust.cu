#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            thrust::host_vector<int> host_in(n);
            thrust::host_vector<int> host_out(n);

            // Copy input into host vectors
            for (int i = 0; i < n; ++i) {
                host_in[i] = idata[i];
            }
            
            // Create device_vectors from host_vectors
            thrust::device_vector<int> dev_in(host_in);
            thrust::device_vector<int> dev_out(host_out);

            timer().startGpuTimer();
            thrust::exclusive_scan(dev_in.begin(), dev_in.end(), dev_out.begin());
            timer().endGpuTimer();

            // Write final results
            for (int i = 0; i < n; i++) {
                odata[i] = dev_out[i];
            }
        }
    }
}
