#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernScan(int N, int offset, int* odata, int* idata) {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;
          if (index >= N)
            return;

          int inValue = idata[index];
          if (index >= offset) 
            odata[index] = idata[index - offset] + inValue;
          else
            odata[index] = inValue;
          
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO
            int offset = 1;
            int logN = ilog2ceil(n);
            int* dev_odata;
            int* dev_idata;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));

            hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            /* Start Timer */
            timer().startGpuTimer();

            for (int d = 0; d < logN; d++, offset *= 2) {
              std::swap(dev_idata, dev_odata);
              kernScan << <fullBlocksPerGrid, blockSize >> > (n, offset, dev_odata, dev_idata);
            }

            timer().endGpuTimer();

            odata[0] = 0;
            hipMemcpy(odata + 1, dev_odata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
        }
    }
}
